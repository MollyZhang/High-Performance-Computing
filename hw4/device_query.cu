#include <stdio.h> 
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>


int main() {
    int nDevices, i;
    hipGetDeviceCount(&nDevices);
    printf("There are %d CUDA devices. \n", nDevices);
    for (i = 0; i < nDevices; i++) {
        hipDeviceProp_t prop;
        hipGetDeviceProperties(&prop, i);
        printf("Device Number: %d\n", i);
        printf("Device name: %s\n", prop.name);
	printf("Total global memory: %u\n", prop.totalGlobalMem);
	printf("Total shared memory per block: %u\n",  prop.sharedMemPerBlock);
    	printf("Total registers per block:     %d\n",  prop.regsPerBlock);
    	printf("Warp size:                     %d\n",  prop.warpSize);
    	printf("Maximum memory pitch:          %u\n",  prop.memPitch);
    	printf("Maximum threads per block:     %d\n",  prop.maxThreadsPerBlock);
    	printf("Clock rate:                    %d\n",  prop.clockRate);
    	printf("Total constant memory:         %u\n",  prop.totalConstMem);
    	printf("Number of multiprocessors:     %d\n",  prop.multiProcessorCount);
        printf("Memory Clock Rate (KHz): %d\n", prop.memoryClockRate);
        printf("Memory Bus Width (bits): %d\n", prop.memoryBusWidth);
        printf("Peak Memory Bandwidth (GB/s): %f\n\n",
            2.0*prop.memoryClockRate*(prop.memoryBusWidth/8)/1.0e6);
  }
}
