#include <stdio.h> 
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>


int main() {
    int nDevices, i;
    hipGetDeviceCount(&nDevices);
    printf("There are %d CUDA devices. \n", nDevices);
    for (i = 0; i < nDevices; i++) {
        hipDeviceProp_t prop;
//        cudaGetDeviceProperties(&prop, i);
//        printf("Device Number: %d\n", i);
//        printf("  Device name: %s\n", prop.name);
//        printf("  Memory Clock Rate (KHz): %d\n",
//            prop.memoryClockRate);
//        printf("  Memory Bus Width (bits): %d\n",
//            prop.memoryBusWidth);
//        printf("  Peak Memory Bandwidth (GB/s): %f\n\n",
//            2.0*prop.memoryClockRate*(prop.memoryBusWidth/8)/1.0e6);
  }
}
